#include "gpen.h"

extern void initialize_io               (void *, const Z, const Z, const Z);
extern void initialize_initial_condition(void *, const Z, const Z, const Z);
extern void initialize_rk_2n            (void *, const Z, const Z, const Z);

static void *f, *g, *h;

static void done(void)
{
  hipFree(f);
  hipFree(g);
  free(h);
}

Q *initialize_modules(const Z nx, const Z ny, const Z nz)
{
  hipError_t err;

  const Z n = nx * ny * nz;

  err = hipMalloc(&f, sizeof(Q) * n);
  if(hipSuccess != err) error(hipGetErrorString(err));

  err = hipMalloc(&g, sizeof(Q) * n);
  if(hipSuccess != err) error(hipGetErrorString(err));

  h = malloc(sizeof(Q) * n);
  if(!h) error("fail to allocate host memory");

  atexit(done);

  initialize_io               (h, nx, ny, nz);
  initialize_initial_condition(h, nx, ny, nz);
  initialize_rk_2n            (g, nx, ny, nz);

  return (Q *)f;
}
