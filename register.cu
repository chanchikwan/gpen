#include "gpen.h"

static void *f, *g, *h;

static void done(void)
{
  hipFree(f);
  hipFree(g);
  free(h);
}

void initialize_modules(const Z nx, const Z ny, const Z nz)
{
  hipError_t err;

  const Z n = nx * ny * nz;

  err = hipMalloc(&f, sizeof(Q) * n);
  if(hipSuccess != err) error(hipGetErrorString(err));

  err = hipMalloc(&g, sizeof(Q) * n);
  if(hipSuccess != err) error(hipGetErrorString(err));

  h = malloc(sizeof(Q) * n);
  if(!h) error("fail to allocate host memory");

  atexit(done);

  /* TODO: set modules */
}
