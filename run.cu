#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include "gpen.h"

Q f0(R x, R y, R z)
{
  Q f;

  x -= 0.5;
  y -= 0.5;
  z -= 0.5;

  f.lnrho = -0.5 * (x * x + y * y + z * z) / 0.01;
  f.ux    = 0.0;
  f.uy    = 0.0;
  f.uz    = 0.0;

  return f;
}

int main(int argc, char *argv[])
{
  const char rotor[] = "-/|\\";

  const R tt = (argc > 1) ? atof(argv[1]) : 1.0;
  const Z nt = (argc > 2) ? atoi(argv[2]) : 100;
  const Z nx = (argc > 3) ? atoi(argv[3]) : 256;
  const Z ny = (argc > 4) ? atoi(argv[4]) :  nx;
  const Z nz = (argc > 5) ? atoi(argv[5]) :  ny;

  const R fo = 0.0;    /* TODO: number of floating-point operations */
  const R dt = 1.0e-3; /* TODO: compute from velocity */

  Q *f = NULL;
  Z  i = 0;
  hipEvent_t t0, t1;
  hipEventCreate(&t0);
  hipEventCreate(&t1);

  printf("G-Pen: reimplementing the pencil code for GPU\n");

  f = initialize_modules(nx, ny, nz);

  initial_condition(f, f0);

  while(output(i++, f) < nt) {
    const Z ns = (Z)ceilf(tt / nt / dt);
    const R ds = tt / nt / ns;

    Z j = 0;
    float ms;

    printf("%4d: %5.2f -> %5.2f: dt ~ %.0e:       ",
           i, ds * ns * (i-1), ds * ns * i, ds);

    hipEventRecord(t0, 0);

    while(j++ < ns) {
      printf("\b\b\b\b\b\b%c %4d", rotor[j%4], j);
      fflush(stdout);

      usleep(10000); /* TODO: time stepping */

      hipDeviceSynchronize();
    }
    hipEventRecord(t1, 0);

    hipEventSynchronize(t1);
    hipEventElapsedTime(&ms, t0, t1); ms /= ns;
    printf("\b\b\b\b\b\b%.3f ms/cycle ~ %.3f GFLOPS\n",
           ms, 1.0e-6 * fo / ms);
  }

  hipEventDestroy(t1);
  hipEventDestroy(t0);
  return 0;
}
