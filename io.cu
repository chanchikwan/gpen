#include <stdio.h>
#include "gpen.h"

static Q *host;
static Z  n;

void initialize_io(void *h, const Z nx, const Z ny, const Z nz)
{
  host = (Q *)h;
  n    = nx * ny * nz;
}

Z output(Z i, const Q *f)
{
  hipError_t err;

  char  name[256];
  FILE *file;

  err = hipMemcpy(host, f, sizeof(Q) * n, hipMemcpyDeviceToHost);
  if(hipSuccess != err) error(hipGetErrorString(err));

  sprintf(name, "%04d.raw", i);
  file = fopen(name, "wb");
  fwrite(host, sizeof(Q), n, file);
  fclose(file);

  return i;
}
