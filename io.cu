#include <stdio.h>
#include "gpen.h"

static R *Host;
static Z Nx, Ny, Nz;
static R Lx, Ly, Lz;

void initialize_io(void *h, const Z nx, const Z ny, const Z nz,
                            const R lx, const R ly, const R lz)
{
  Host = (R *)h;

  Nx = nx;
  Ny = ny;
  Nz = nz;

  Lx = lx;
  Ly = ly;
  Lz = lz;
}

Z output(Z i, const R *f)
{
  hipError_t err;

  const Z ndata  = Nx * Ny * Nz;
  const Z hghost = (Nx * Ny + Ny * Nz + Nz * Nx) * RADIUS;
  const Z ntotal = ndata + 2 * hghost;

  char  name[256];
  FILE *file;

  Z h;

  for(h = 0; h < N_VAR; ++h) {
    err = hipMemcpy(Host + h * ndata, f + hghost + h * ntotal,
                     sizeof(R) * ndata, hipMemcpyDeviceToHost);
    if(hipSuccess != err) error(hipGetErrorString(err));
  }

  sprintf(name, "%04d.raw", i);
  file = fopen(name, "wb");
  fwrite(&Lx,  sizeof(R), 1, file);
  fwrite(&Ly,  sizeof(R), 1, file);
  fwrite(&Lz,  sizeof(R), 1, file);
  fwrite(&Nx,  sizeof(Z), 1, file);
  fwrite(&Ny,  sizeof(Z), 1, file);
  fwrite(&Nz,  sizeof(Z), 1, file);
  fwrite(&h,   sizeof(Z), 1, file); /* after the for-loop, h == N_VAR */
  fwrite(Host, sizeof(R), ndata * N_VAR, file);
  fclose(file);

  return i;
}
